#include "hip/hip_runtime.h"
/**
 * MatrixKernel.cu
 *
 *  Created on: Jul 30, 2013
 *      Author: Zahari Dichev <zaharidichev@gmail.com>
 */

#include "stdio.h"
#include "../../misc/Macros.h"
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline __device__ int getID() {
	return blockIdx.x * blockDim.x + threadIdx.x;
}

__global__ void MatrixMulKernel(float* d_M, float* d_N, float* d_P, int Width, int tileWidth, int totalThrs) {

	int tID = getID();

	if (tID != totalThrs - 1) {
		int start_row = (tID * tileWidth);
		int end_row = (start_row + tileWidth);
		int start_col = (tID * tileWidth);
		int end_col = (start_col + tileWidth);

		for (int row = start_row; row < end_row; row++) {
			for (int col = start_col; col < end_col; col++) {
				float P_val = 0;
				for (int k = 0; k < Width; ++k) {
					float M_elem = d_M[row * Width + k];
					float N_elem = d_N[k * Width + col];
					P_val += M_elem * N_elem;
				}
				d_P[row * Width + col] = P_val;
			}
		}
	}

}

extern "C" void startMMKernel(size_t threads, size_t blocks, float* d_M, float* d_N, float* d_P, int mtrxWidth, int tileWidth, int totalThrs,
		hipStream_t stream) {
	MatrixMulKernel<<<blocks, threads, 0, stream>>>(d_M, d_N, d_P, mtrxWidth, tileWidth, totalThrs);
	gpuErrchk(hipGetLastError());

}

extern "C" hipFuncAttributes getMMKernelProperties() {
	hipFuncAttributes attributes;
	hipFuncGetAttributes(&attributes, reinterpret_cast<const void*>(MatrixMulKernel));
	return attributes;
}
